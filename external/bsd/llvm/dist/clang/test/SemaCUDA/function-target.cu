#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

__host__ void h1h(void);
__device__ void h1d(void); // expected-note {{candidate function not viable: call to __device__ function from __host__ function}}
__host__ __device__ void h1hd(void);
__global__ void h1g(void);

struct h1ds { // expected-note {{requires 1 argument}}
  __device__ h1ds(); // expected-note {{candidate constructor not viable: call to __device__ function from __host__ function}}
};

__host__ void h1(void) {
  h1h();
  h1d(); // expected-error {{no matching function}}
  h1hd();
  h1g<<<1, 1>>>();
  h1ds x; // expected-error {{no matching constructor}}
}

__host__ void d1h(void); // expected-note {{candidate function not viable: call to __host__ function from __device__ function}}
__device__ void d1d(void);
__host__ __device__ void d1hd(void);
__global__ void d1g(void); // expected-note {{'d1g' declared here}}

__device__ void d1(void) {
  d1h(); // expected-error {{no matching function}}
  d1d();
  d1hd();
  d1g<<<1, 1>>>(); // expected-error {{reference to __global__ function 'd1g' in __device__ function}}
}

// Expected 0-1 as in one of host/device side compilation it is an error, while
// not in the other
__host__ void hd1h(void); // expected-note 0-1 {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
__device__ void hd1d(void); // expected-note 0-1 {{candidate function not viable: call to __device__ function from __host__ __device__ function}}
__host__ void hd1hg(void);
__device__ void hd1dg(void);
#ifdef __CUDA_ARCH__
__host__ void hd1hig(void); // expected-note {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
#else
__device__ void hd1dig(void); // expected-note {{candidate function not viable: call to __device__ function from __host__ __device__ function}}
#endif
__host__ __device__ void hd1hd(void);
__global__ void hd1g(void); // expected-note {{'hd1g' declared here}}

__host__ __device__ void hd1(void) {
  // Expected 0-1 as in one of host/device side compilation it is an error,
  // while not in the other
  hd1d(); // expected-error 0-1 {{no matching function}}
  hd1h(); // expected-error 0-1 {{no matching function}}

  // No errors as guarded
#ifdef __CUDA_ARCH__
  hd1d();
#else
  hd1h();
#endif

  // Errors as incorrectly guarded
#ifndef __CUDA_ARCH__
  hd1dig(); // expected-error {{no matching function}}
#else
  hd1hig(); // expected-error {{no matching function}}
#endif

  hd1hd();
  hd1g<<<1, 1>>>(); // expected-error {{reference to __global__ function 'hd1g' in __host__ __device__ function}}
}
