
#include <hip/hip_runtime.h>
// RUN: c-index-test -test-load-source all -x cuda %s | FileCheck %s

__attribute__((device)) void f_device();
__attribute__((global)) void f_global();
__attribute__((constant)) int* g_constant;
__attribute__((shared)) float *g_shared;
__attribute__((host)) void f_host();

// CHECK:       attributes-cuda.cu:3:30: FunctionDecl=f_device:3:30
// CHECK-NEXT:  attributes-cuda.cu:3:16: attribute(device)
// CHECK:       attributes-cuda.cu:4:30: FunctionDecl=f_global:4:30
// CHECK-NEXT:  attributes-cuda.cu:4:16: attribute(global)
// CHECK:       attributes-cuda.cu:5:32: VarDecl=g_constant:5:32 (Definition)
// CHECK-NEXT:  attributes-cuda.cu:5:16: attribute(constant)
// CHECK:       attributes-cuda.cu:6:32: VarDecl=g_shared:6:32 (Definition)
// CHECK-NEXT:  attributes-cuda.cu:6:16: attribute(shared)
// CHECK:       attributes-cuda.cu:7:28: FunctionDecl=f_host:7:28
// CHECK-NEXT:  attributes-cuda.cu:7:16: attribute(host)
