#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -triple nvptx-unknown-unknown -fcuda-is-device -emit-llvm -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

#define MAX_THREADS_PER_BLOCK 256
#define MIN_BLOCKS_PER_MP     2

// Test both max threads per block and Min cta per sm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel1()
{
}
}

// CHECK: !{{[0-9]+}} = !{void ()* @Kernel1, !"maxntidx", i32 256}
// CHECK: !{{[0-9]+}} = !{void ()* @Kernel1, !"minctasm", i32 2}

// Test only max threads per block. Min cta per sm defaults to 0, and
// CodeGen doesn't output a zero value for minctasm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK )
Kernel2()
{
}
}

// CHECK: !{{[0-9]+}} = !{void ()* @Kernel2, !"maxntidx", i32 256}
