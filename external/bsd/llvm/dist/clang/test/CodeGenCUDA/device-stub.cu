#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -emit-llvm %s -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// Test that we build the correct number of calls to hipSetupArgument followed
// by a call to hipLaunchByPtr.

// CHECK: define{{.*}}kernelfunc
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipLaunchByPtr
__global__ void kernelfunc(int i, int j, int k) {}
